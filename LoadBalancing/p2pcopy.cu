// 
// Adapted from Paulius Micikevicius (pauliusm@nvidia.com)
//


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define	NUM_GPUS	2

void process_error( const hipError_t &error, char *string=0, bool verbose=false )
{
	if( error != hipSuccess || verbose )
	{
		if( string )
			printf( string );
		printf( ": %s\n", hipGetErrorString( error ) );
	}

	if( error != hipSuccess )
		exit(-1);
}

int main( int argc, char *argv[] )
{
	size_t num_bytes = 16*1024*1024;
	int nreps = 10;
	int gpu_0 = 0;
	int gpu_1 = 1;

	if( argc >= 2 )
		num_bytes = (size_t)( atoi( argv[1] ) * 1024*1024 );
	if( argc >= 3 )
		nreps = atoi( argv[2] );
	if( argc >= 4 )
		gpu_0 = atoi( argv[3] );
	if( argc >= 5 )
		gpu_1 = atoi( argv[4] );

	hipError_t error = hipSuccess;
	
	hipDeviceProp_t gpu_prop;
	hipGetDeviceProperties( &gpu_prop, gpu_0 );
	printf("GPU 0: %s\n", gpu_prop.name  );
	hipGetDeviceProperties( &gpu_prop, gpu_1 );
	printf("GPU 1: %s\n", gpu_prop.name  );
	
	void *d_a[4] = {0, 0, 0, 0};
	void *d_b[4] = {0, 0, 0, 0};

	hipSetDevice( gpu_0 );
	error = hipMalloc( &d_a[0], num_bytes );
	process_error( error, "allocate a on GPU 0" );
	error = hipMalloc( &d_b[0], num_bytes );
	process_error( error, "allocate b on GPU 0" );
	error = hipDeviceEnablePeerAccess( gpu_1, 0 );
	process_error( error, "enable GPU 0 to access GPU 1's memory" );

	hipSetDevice( gpu_1 );
	error = hipMalloc( &d_a[1], num_bytes );
	process_error( error, "allocate a on GPU 1" );
	error = hipMalloc( &d_b[1], num_bytes );
	process_error( error, "allocate b on GPU 1" );
	error = hipDeviceEnablePeerAccess( gpu_0, 0 );
	process_error( error, "enable GPU 1 to access GPU 0's memory" );
	
	hipSetDevice( gpu_0 );

	float elapsed_time_ms = 0.f;
	float throughput_gbs  = 0.f;
	hipEvent_t start, stop;
	hipEventCreate( &start );
	hipEventCreate( &stop );
	hipStream_t stream_on_gpu_0, stream_on_gpu_1;

        hipSetDevice( gpu_0 );
        hipStreamCreate( &stream_on_gpu_0 );
        hipSetDevice( gpu_1 );
        hipStreamCreate( &stream_on_gpu_1 );

	hipSetDevice( gpu_0 );

	///////////////////////////
	// pull copy
	//
	hipEventRecord( start, 0 );
	for( int i=0; i<nreps; i++ ) {
		hipMemcpyPeerAsync( d_b[0], gpu_0, d_b[1], gpu_1, num_bytes, stream_on_gpu_0 );
	}
	error = hipStreamSynchronize(stream_on_gpu_0);
	hipEventRecord( stop, 0 );
	error = hipDeviceSynchronize();
	process_error( error, "sync after pull copy" );
		
	error = hipEventElapsedTime( &elapsed_time_ms, start, stop );
	process_error( error, "get event elapsed time" );
	elapsed_time_ms /= nreps;
	throughput_gbs = num_bytes * 1e-6f / elapsed_time_ms;
	printf( "%d ->%d: %7.2f %7.2f\n", gpu_1, gpu_0, elapsed_time_ms, throughput_gbs );

	///////////////////////////
	// push copy
	//
	hipEventRecord( start, 0 );
	for( int i=0; i<nreps; i++ ) {
		hipMemcpyPeerAsync( d_a[1], gpu_1, d_a[0], gpu_0, num_bytes, stream_on_gpu_0 );
	}
	hipEventRecord( stop, 0 );
	error = hipDeviceSynchronize();
	process_error( error, "sync after push copy" );
		
	error = hipEventElapsedTime( &elapsed_time_ms, start, stop );
	process_error( error, "get event elapsed time" );
	elapsed_time_ms /= nreps;
	throughput_gbs = num_bytes * 1e-6f / elapsed_time_ms;
	printf( "%d ->%d: %7.2f %7.2f\n", gpu_0, gpu_1, elapsed_time_ms, throughput_gbs );


	///////////////////////////
	// exchange with sync
	//
	hipEventRecord( start, 0 );
	for( int i=0; i<nreps; i++ ) {
		hipMemcpyPeerAsync( d_a[1], gpu_1, d_a[0], gpu_0, num_bytes, stream_on_gpu_0 );
		hipMemcpyPeerAsync( d_b[0], gpu_0, d_b[1], gpu_1, num_bytes, stream_on_gpu_1 );
		hipDeviceSynchronize();
	}
	hipEventRecord( stop, 0 );
	error = hipDeviceSynchronize();
	process_error( error, "sync after exchange" );
		
	error = hipEventElapsedTime( &elapsed_time_ms, start, stop );
	process_error( error, "get event elapsed time" );
	elapsed_time_ms /= nreps;
	throughput_gbs = num_bytes * 2e-6f / elapsed_time_ms;
	printf( "%d<->%d: %7.2f %7.2f\n", gpu_0, gpu_1, elapsed_time_ms, throughput_gbs );

	///////////////////////////
	// exchange without sync
	//
	hipEventRecord( start, 0 );
	for( int i=0; i<nreps; i++ ) {
		hipMemcpyPeerAsync( d_a[1], gpu_1, d_a[0], gpu_0, num_bytes, stream_on_gpu_0 );
		hipMemcpyPeerAsync( d_b[0], gpu_0, d_b[1], gpu_1, num_bytes, stream_on_gpu_1 );
	}
	hipEventRecord( stop, 0 );
	error = hipDeviceSynchronize();
	process_error( error, "sync after exchange" );
		
	error = hipEventElapsedTime( &elapsed_time_ms, start, stop );
	process_error( error, "get event elapsed time" );
	elapsed_time_ms /= nreps;
	throughput_gbs = num_bytes * 2e-6f / elapsed_time_ms;
	printf( "%d<->%d: %7.2f %7.2f\n", gpu_0, gpu_1, elapsed_time_ms, throughput_gbs );


	
	hipSetDevice( gpu_0 );
	error = hipFree( d_a[0] );
	process_error( error, "free memory on GPU 0" );
	error = hipDeviceReset();
	process_error( error, "reset GPU 0" );

	hipSetDevice( gpu_1 );
	error = hipFree( d_a[1] );
	process_error( error, "free memory on GPU 1" );
	error = hipDeviceReset();
	process_error( error, "reset GPU 1" );
	
	printf("CUDA: %s\n", hipGetErrorString( hipGetLastError() ) );

	return 0;
}
