#include "hip/hip_runtime.h"
#include "CudaKernelsSimpleBloom.h"

#define NUM_THREADS 256

static __constant__ CudaSBFInputElem constInput[MAX_ATTR_NUM];

static __device__ bool cuda_strcmp(char *s1, char *s2) {
	for ( ; *s1==*s2; ++s1, ++s2) {
		if (*s1=='\0') return true;
	}
	return false;
}

static __device__ bool cuda_prefix(char *s1, char *s2) {
	for ( ; *s1==*s2; ++s1, ++s2) {
		if (*(s2+1)=='\0') return true;
	}
	return false;
}

static __device__ bool cuda_substr(char *s1, char *s2) {
	int size1 = 0;
	int size2 = 0;
	while (s1[size1]!='\0') size1++;
	while (s2[size2]!='\0') size2++;
	if (size1==size2) return cuda_strcmp(s1, s2);
	if (size1<size2) return false;
	for (int i=0; i<size1-size2+1; i++) {
		bool failed = false;
		for (int j=0; j<size2; j++) {
			if (s1[i+j-1]!=s2[j]) {
				failed = true;
				break;
			}
		}
		if (! failed) return true;
	}
	return false;
}

static __global__ void cleanCounters(unsigned int *filtersCount, unsigned char *interfaces, const int numFilters, const int numInterfaces) {
	int pos = blockIdx.x*blockDim.x+threadIdx.x;
	// initialize interfaces and filtersCount
	if (pos<numInterfaces) interfaces[pos] = 0;
	while(pos<numFilters) {
		filtersCount[pos] = 0;
		pos = pos + gridDim.x*blockDim.x;
	}
}

template <int byteId>
static __global__ void evalConstraint(unsigned int *filtersCount, const FilterInfo *filterInfo, unsigned char *interfaces, const int numFilters, const int numInterfaces, const BFilter inputBF) {
	int constraintsIndex = blockIdx.x*blockDim.x+threadIdx.x;
	if (constraintsIndex>=constInput[blockIdx.y].numConstraints) return;
	CudaSBFInputElem inputElem = constInput[blockIdx.y];
	if((inputBF & inputElem.filterNames[constraintsIndex])!=inputElem.filterNames[constraintsIndex]) return;
	CudaValue val = inputElem.value;
	Op constrOp = inputElem.constrOp[constraintsIndex];
	if (val.type==INT) {
		IntCudaConstraint constrVal = ((IntCudaConstraint *)inputElem.constrVal)[constraintsIndex];
		if ((constrOp==EQ && val.intVal!=constrVal.value) ||
				(constrOp==LT && val.intVal>=constrVal.value) ||
				(constrOp==GT && val.intVal<=constrVal.value) ||
				(constrOp==DF && val.intVal==constrVal.value)) return;
	} else {
		StringCudaConstraint constrVal = ((StringCudaConstraint *)inputElem.constrVal)[constraintsIndex];
		if ((constrOp==EQ && !cuda_strcmp(val.stringVal, constrVal.value)) ||
				(constrOp==DF &&  cuda_strcmp(val.stringVal, constrVal.value)) ||
				(constrOp==PF && !cuda_prefix(val.stringVal, constrVal.value)) ||
				(constrOp==IN && !cuda_substr(val.stringVal, constrVal.value))) return;
	}
	int filterIndex = inputElem.filterIdx[constraintsIndex];
	int count;
	if (byteId==1) {
		count = atomicAdd(&filtersCount[filterIndex], 1);
	} else if (byteId==2) {
		count = atomicAdd(&filtersCount[filterIndex], 256);
		count = (count >> 8) & 0xF;
	} else if (byteId==3) {
		count = atomicAdd(&filtersCount[filterIndex], 65536);
		count = (count >> 16) & 0xF;
	} else if (byteId==4) {
		count = atomicAdd(&filtersCount[filterIndex], 16777216);
		count = (count >> 24) & 0xF;
	}
	if (count+1==filterInfo[filterIndex].numConstraints) {
		interfaces[filterInfo[filterIndex].interface] = 1;
	}
}

static inline void add(BFilter &bf1, const char* name) {
	bf1 = bf1 | 1u << ((name[0]+name[1])%(sizeof(BFilter)*8));
	bf1 = bf1 | 1u << ((name[2]+name[3])%(sizeof(BFilter)*8));
	bf1 = bf1 | 1u << ((name[4]+name[5])%(sizeof(BFilter)*8));
	bf1 = bf1 | 1u << ((name[6]+name[7])%(sizeof(BFilter)*8));
}

CudaKernelsSimpleBloom::CudaKernelsSimpleBloom() {
	currentByteId = 1;
	numInterfaces = 0;
	numFilters = 0;
	consolidated = false;
	hostToDeviceCopyTime = 0;
	execTime = 0;
	deviceToHostCopyTime = 0;
}

CudaKernelsSimpleBloom::~CudaKernelsSimpleBloom() {
	if (consolidated) {
		for (map<string_t, void *>::iterator it=nameDeviceConstrVal.begin(); it!=nameDeviceConstrVal.end(); ++it) {
			void *constrPtr = it->second;
			hipFree(constrPtr);
		}
		for (map<string_t, Op *>::iterator it=nameDeviceConstrOp.begin(); it!=nameDeviceConstrOp.end(); ++it) {
			Op *constrPtr = it->second;
			hipFree(constrPtr);
		}
		for (map<string_t, int *>::iterator it=nameDeviceFilterIdx.begin(); it!=nameDeviceFilterIdx.end(); ++it) {
			int *filterIdxPtr = it->second;
			hipFree(filterIdxPtr);
		}
		for (map<string_t, BFilter *>::iterator it=nameDeviceFilterNames.begin(); it!=nameDeviceFilterNames.end(); ++it) {
			BFilter *filterNamePtr = it->second;
			hipFree(filterNamePtr);
		}
		hipHostFree(hostInput);
		hipFree(currentFiltersCount);
		hipFree(filtersInfo);
		hipFree(interfacesDevice);
		hipHostFree(interfacesHost);
	}
	for (map<int, set<CudaFilter *> >::iterator it=hostFilters.begin(); it!=hostFilters.end(); ++it) {
		for (set<CudaFilter *>::iterator it2=it->second.begin(); it2!=it->second.end(); ++it2) {
			CudaFilter *filter = *it2;
			delete filter;
		}
	}
}

void CudaKernelsSimpleBloom::ifConfig(int interfaceId, set<CudaFilter *> &filters) {
	// record the set of filters associated to this interface
	hostFilters.insert(make_pair(interfaceId, filters));

	// update the numConstraints and nameType data structures (to be used at consolidate time)
	for (set<CudaFilter *>::iterator it=filters.begin(); it!=filters.end(); ++it) {
		CudaFilter *filter = *it;
		for (int i=0; i<filter->numConstraints; i++) {
			string_t nameStr = filter->constraints[i].name;
			map<string_t, int>::iterator it=numConstraints.find(nameStr);
			if (it==numConstraints.end()) {
				numConstraints.insert(make_pair(nameStr, 1));
			} else {
				it->second++;
			}
			map<string_t, Type>::iterator it1=nameType.find(nameStr);
			if (it1==nameType.end()) {
				nameType.insert(make_pair(nameStr, filter->constraints[i].value.type));
			}
		}
		numFilters++;
	}
}

void CudaKernelsSimpleBloom::consolidate() {
	// allocate memory on device and host
	int e = 0;
	int allocSize = 0;
	numInterfaces = hostFilters.size();
	allocSize += sizeof(CudaSBFInputElem)*MAX_ATTR_NUM;  // allocated into constant memory (see static variable at the beginning of file)
	e += hipHostMalloc((void**) &hostInput, (size_t) sizeof(CudaSBFInputElem)*MAX_ATTR_NUM);
	e += hipMalloc((void**) &interfacesDevice, (size_t) sizeof(unsigned char)*numInterfaces);
	allocSize += sizeof(unsigned char)*numInterfaces;
	e += hipHostMalloc((void**) &interfacesHost, (size_t) sizeof(unsigned char)*numInterfaces);
	map<string_t, int> currentNumConstraints;
	map<string_t, void *> nameHostConstrVal;
	map<string_t, Op *> nameHostConstrOp;
	map<string_t, int *> nameHostFilterIdx;
	map<string_t, BFilter *> nameHostFilterNames;
	for (map<string_t, int>::iterator it=numConstraints.begin(); it!=numConstraints.end(); ++it) {
		string_t name = it->first;
		int num = it->second;
		void *constrValPtr, *hostConstrValPtr;
		if(nameType[name]==INT) {
			e += hipMalloc((void**) &constrValPtr, (size_t) sizeof(IntCudaConstraint)*num);
			hostConstrValPtr = malloc(sizeof(IntCudaConstraint)*num);
			allocSize += sizeof(IntCudaConstraint)*num;
		} else {
			e += hipMalloc((void**) &constrValPtr, (size_t) sizeof(StringCudaConstraint)*num);
			hostConstrValPtr = malloc(sizeof(StringCudaConstraint)*num);
			allocSize += sizeof(StringCudaConstraint)*num;
		}
		nameDeviceConstrVal.insert(make_pair(name, constrValPtr));
		nameHostConstrVal.insert(make_pair(name, hostConstrValPtr));
		Op *constrOpPtr, *hostConstrOpPtr;
		e+= hipMalloc((void**) &constrOpPtr, (size_t) sizeof(Op)*num);
		hostConstrOpPtr = (Op *)malloc(sizeof(Op)*num);
		allocSize += sizeof(Op)*num;
		nameDeviceConstrOp.insert(make_pair(name, constrOpPtr));
		nameHostConstrOp.insert(make_pair(name, hostConstrOpPtr));
		currentNumConstraints.insert(make_pair(name, 0));
		int *filterIdxPtr, *hostFilterIdxPtr;
		e+= hipMalloc((void**) &filterIdxPtr, (size_t) sizeof(int)*num);
		hostFilterIdxPtr = (int *)malloc(sizeof(int)*num);
		allocSize += sizeof(int)*num;
		nameDeviceFilterIdx.insert(make_pair(name, filterIdxPtr));
		nameHostFilterIdx.insert(make_pair(name, hostFilterIdxPtr));
		BFilter *filterNamesPtr, *hostFilterNamesPtr;
		e+= hipMalloc((void**) &filterNamesPtr, (size_t) sizeof(BFilter)*num);
		hostFilterNamesPtr = (BFilter *)malloc(sizeof(int)*num);
		allocSize += sizeof(BFilter)*num;
		nameDeviceFilterNames.insert(make_pair(name, filterNamesPtr));
		nameHostFilterNames.insert(make_pair(name, hostFilterNamesPtr));
	}
	e += hipMalloc((void**) &currentFiltersCount, (size_t) sizeof(unsigned int)*numFilters);
	allocSize += sizeof(unsigned int)*numFilters;
	e += hipMalloc((void**) &filtersInfo, (size_t) sizeof(FilterInfo)*numFilters);
	allocSize += sizeof(FilterInfo)*numFilters;
	if (e>0) {
		cerr << " Allocation error " << e << endl;
		exit(1);
	}

	// initialize the nameHostConstrVal, nameHostConstrOp, nameHostFilterIdx, nameHostFilterNames, and hostFiltersInfo structures
	// (to be copied into the corresponding structures in device later)
	int filterId = 0;
	FilterInfo *hostFiltersInfo = (FilterInfo *) malloc(sizeof(FilterInfo)*numFilters);
	for (map<int, set<CudaFilter *> >::iterator it=hostFilters.begin(); it!=hostFilters.end(); ++it) {
		int interfaceId = it->first;
		for (set<CudaFilter *>::iterator it2=it->second.begin(); it2!=it->second.end(); ++it2) {
			CudaFilter *filter = *it2;
			BFilter bf=0;
			for (int i=0; i<filter->numConstraints; i++) {
				add(bf,filter->constraints[i].name);
			}
			for (int i=0; i<filter->numConstraints; i++) {
				string_t name = filter->constraints[i].name;
				int writingIndex = currentNumConstraints[name];
				currentNumConstraints[name] = writingIndex+1;
				Op *hostConstrOpPtr = nameHostConstrOp[name];
				hostConstrOpPtr[writingIndex] = filter->constraints[i].op;
				if(nameType[name]==INT) {
					IntCudaConstraint *hostConstrValPtr = (IntCudaConstraint *)nameHostConstrVal[name];
					hostConstrValPtr[writingIndex].value = filter->constraints[i].value.intVal;
				} else {
					StringCudaConstraint *hostConstrValPtr = (StringCudaConstraint *)nameHostConstrVal[name];
					memcpy(hostConstrValPtr[writingIndex].value, filter->constraints[i].value.stringVal, STRING_VAL_LEN);
				}
				int *hostFilterIdxPtr = nameHostFilterIdx[name];
				hostFilterIdxPtr[writingIndex] = filterId;
				BFilter *hostFilterNamesPtr = nameHostFilterNames[name];
				hostFilterNamesPtr[writingIndex] = bf;
			}
			hostFiltersInfo[filterId].numConstraints = filter->numConstraints;
			hostFiltersInfo[filterId].interface = interfaceId;
			filterId++;
		}
	}

	// initialize the device memory
	for (map<string_t, void *>::iterator it=nameHostConstrVal.begin(); it!=nameHostConstrVal.end(); ++it) {
		string_t name = it->first;
		void *host = it->second;
		void *device = nameDeviceConstrVal[name];
		int size = numConstraints[name];
		if(nameType[name]==INT) {
			e += hipMemcpy(device, host, sizeof(IntCudaConstraint)*size, hipMemcpyHostToDevice);
		} else {
			e += hipMemcpy(device, host, sizeof(StringCudaConstraint)*size, hipMemcpyHostToDevice);
		}
		hipDeviceSynchronize();
		free(host);
	}
	for (map<string_t, Op *>::iterator it=nameHostConstrOp.begin(); it!=nameHostConstrOp.end(); ++it) {
		string_t name = it->first;
		Op *host = it->second;
		Op *device = nameDeviceConstrOp[name];
		int size = numConstraints[name];
		e += hipMemcpy(device, host, sizeof(Op)*size, hipMemcpyHostToDevice);
		hipDeviceSynchronize();
		free(host);
	}
	for (map<string_t, int *>::iterator it=nameHostFilterIdx.begin(); it!=nameHostFilterIdx.end(); ++it) {
		string_t name = it->first;
		int *host = it->second;
		int *device = nameDeviceFilterIdx[name];
		int size = numConstraints[name];
		e += hipMemcpy(device, host, sizeof(int)*size, hipMemcpyHostToDevice);
		hipDeviceSynchronize();
		free(host);
	}
	for (map<string_t, BFilter *>::iterator it=nameHostFilterNames.begin(); it!=nameHostFilterNames.end(); ++it) {
		string_t name = it->first;
		BFilter *host = it->second;
		BFilter *device = nameDeviceFilterNames[name];
		int size = numConstraints[name];
		e += hipMemcpy(device, host, sizeof(BFilter)*size, hipMemcpyHostToDevice);
		hipDeviceSynchronize();
		free(host);
	}
	e += hipMemcpy(filtersInfo, hostFiltersInfo, (size_t) sizeof(FilterInfo)*numFilters, hipMemcpyHostToDevice);
	hipMemset(currentFiltersCount, 0, (size_t) sizeof(unsigned int)*numFilters);
	hipMemset(interfacesDevice, 0, (size_t) sizeof(unsigned char)*numInterfaces);
	hipDeviceSynchronize();
	consolidated = true;
	if (e>0) {
		cerr << " Memcpy error " << e << " during consolidation " <<  endl;
		exit(1);
	}
	free(hostFiltersInfo);

	// set up the runtime to optimize performance
	hipDeviceSetCacheConfig(hipFuncCachePreferL1);

	int totConstr=0;
	for(map<string_t,int>::iterator it=numConstraints.begin(); it!=numConstraints.end(); ++it) {
		totConstr+=it->second;
	}
	cout << endl << " ### " << totConstr << " constraints allocated ### " << endl;
	cout << endl << " ### " << allocSize << " bytes allocated on device ### " << endl;
	cout << endl << "#####################" << endl;
}

void CudaKernelsSimpleBloom::getStats(double &hToD, double &exec, double &dToH) {
	hToD = hostToDeviceCopyTime;
	exec = execTime;
	dToH = deviceToHostCopyTime;
}

#if STATS==1
void CudaKernelsSimpleBloom::processMessage(CudaOutbox *outbox) {
	Timer t;
	t.start();
	BFilter messageNames = copyMsgToDevice(outbox->message);
	//hipDeviceSynchronize(); // TODO: remove
	hostToDeviceCopyTime += t.stop();
	t.start();
	computeResults(messageNames);
	//hipDeviceSynchronize(); // TODO: remove
	execTime += t.stop();
	t.start();
	getMatchingInterfaces(outbox->outgoingInterfaces);
	//hipDeviceSynchronize(); // TODO: remove
	deviceToHostCopyTime += t.stop();
}
#elif STATS==0
void CudaKernelsSimpleBloom::processMessage(CudaOutbox *outbox) {
	BFilter messageNames = copyMsgToDevice(outbox->message);
	computeResults(messageNames);
	getMatchingInterfaces(outbox->outgoingInterfaces);
}
#endif

BFilter CudaKernelsSimpleBloom::copyMsgToDevice(CudaMessage *message) {
	BFilter bf=0;
	int dest = 0;
	for (int i=0; i<message->numAttributes; i++) {
		add(bf, message->attributes[i].name);
		string_t name = message->attributes[i].name;
		hostInput[dest].constrVal = nameDeviceConstrVal[name];
		hostInput[dest].constrOp = nameDeviceConstrOp[name];
		hostInput[dest].filterIdx = nameDeviceFilterIdx[name];
		hostInput[dest].filterNames = nameDeviceFilterNames[name];
		hostInput[dest].numConstraints = numConstraints[name];
		hostInput[dest].value = message->attributes[i].value;
		dest++;
	}
	numValues = dest;
	if (dest>0) {
		int e = 0;
		e += hipMemcpyToSymbolAsync(HIP_SYMBOL(constInput), hostInput, (size_t) sizeof(CudaSBFInputElem)*numValues);
		if (e>0) {
			cerr << " Memcpy error " << e << " during message processing " <<  endl;
			exit(1);
		}
	}
	return bf;
}

void CudaKernelsSimpleBloom::computeResults(BFilter messageNames) {
	int maxNumConstr = 0;
	for(int i=0; i<numValues; i++) {
		if(maxNumConstr<hostInput[i].numConstraints) maxNumConstr=hostInput[i].numConstraints;
	}
	switch (currentByteId) {
	case 1:
		evalConstraint<1><<<dim3(maxNumConstr/NUM_THREADS+1,numValues), NUM_THREADS>>>(currentFiltersCount, filtersInfo, interfacesDevice, numFilters, numInterfaces, messageNames);
		break;
	case 2:
		evalConstraint<2><<<dim3(maxNumConstr/NUM_THREADS+1,numValues), NUM_THREADS>>>(currentFiltersCount, filtersInfo, interfacesDevice, numFilters, numInterfaces, messageNames);
		break;
	case 3:
		evalConstraint<3><<<dim3(maxNumConstr/NUM_THREADS+1,numValues), NUM_THREADS>>>(currentFiltersCount, filtersInfo, interfacesDevice, numFilters, numInterfaces, messageNames);
		break;
	case 4:
		evalConstraint<4><<<dim3(maxNumConstr/NUM_THREADS+1,numValues), NUM_THREADS>>>(currentFiltersCount, filtersInfo, interfacesDevice, numFilters, numInterfaces, messageNames);
		break;
	}
}

void CudaKernelsSimpleBloom::getMatchingInterfaces(set<int> &results) {
	int e = hipMemcpyAsync(interfacesHost, interfacesDevice, (size_t) sizeof(unsigned char)*numInterfaces, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	//hipMemsetAsync(currentFiltersCount, 0, (size_t) sizeof(unsigned int)*numFilters);
	if (currentByteId!=4) {
		hipMemsetAsync(interfacesDevice, 0, (size_t) sizeof(unsigned char)*numInterfaces);
	} else {
		cleanCounters<<<numFilters/2048, NUM_THREADS>>>(currentFiltersCount, interfacesDevice, numFilters, numInterfaces);
	}
	currentByteId++;
	if (currentByteId>4) currentByteId = 1;
	if (e>0) {
		cerr << " Memcpy error " << e << " while copying matching interfaces " <<  endl;
		exit(1);
	}
	for (int i=0; i<numInterfaces; i++) {
		if (interfacesHost[i]!=0) {
			results.insert(i);
		}
	}
}
